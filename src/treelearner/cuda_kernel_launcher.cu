
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "cuda_kernel_launcher.h"
#include <cuda_runtime.h>
#include <cstdio>
#include <LightGBM/utils/log.h>

using namespace LightGBM;

void cuda_histogram(
                data_size_t     leaf_num_data,
                data_size_t     num_data,
                bool            use_all_features,
                bool            is_constant_hessian,
                int             num_workgroups,
                cudaStream_t    stream,
                uint8_t*        arg0,
                uint8_t*        arg1,
                data_size_t     arg2,
                data_size_t*    arg3,
                data_size_t     arg4,
                score_t*        arg5,
                score_t*        arg6,
                score_t         arg6_const,
                char*           arg7,
                volatile int*   arg8,
                void*		arg9,
                size_t          exp_workgroups_per_feature) {


 if (leaf_num_data == num_data) {
 
   if (use_all_features){
     if (!is_constant_hessian) {
       histogram256_fulldata<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         static_cast<float*>(arg6),
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
     else {
       histogram256_fulldata<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         arg6_const,
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
   }
   else{   
   if (!is_constant_hessian) { 
     histogram256_fulldata<<<num_workgroups, 256, 0, stream>>>(
       arg0,
       arg1,
       arg2,
       reinterpret_cast<const uint*>(arg3),
       arg4,
       arg5,
       static_cast<float*>(arg6),
       arg7,
       arg8,
       static_cast<acc_type*>(arg9),
       exp_workgroups_per_feature);
   }
   else { 
     histogram256_fulldata<<<num_workgroups, 256, 0, stream>>>(
       arg0,
       arg1,
       arg2,
       reinterpret_cast<const uint*>(arg3),
       arg4,
       arg5,
       arg6_const, 
       arg7,
       arg8,
       static_cast<acc_type*>(arg9),
       exp_workgroups_per_feature);
   }
  }
 }
 else {
   if (use_all_features) {
     // seems all features is always enabled, so this should be the same as fulldata
     if (!is_constant_hessian) { 

       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         static_cast<float*>(arg6),
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
     else { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         arg6_const, 
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     } 
   }
   else {
     if (!is_constant_hessian) { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         static_cast<float*>(arg6),
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
     else { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         arg6_const, 
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
   }
 }
}

#endif // USE_CUDA
